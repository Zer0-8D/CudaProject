
#include "hip/hip_runtime.h"

#include <iostream>
#include <string>

#include <stdio.h>

void RGB2GSCALE_CPU(const uchar4* const rbg_img, unsigned char *const grsc_img, const size_t rowCount, const size_t colCount);

__global__ void grayscaleKernel(const uchar4* const rbg_img, unsigned char *const grsc_img)
{
	int displacement = threadIdx.x + blockIdx.x * blockDim.x;

	const uchar4 color = rbg_img[displacement];
	const float channelSum = (.299f * color.x) + (.587f * color.y) + (.114f * color.z);
	grsc_img[displacement] = channelSum;

}

int main()
{
    
	hipError_t cudaStatus;


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for grayscale CUDA function
hipError_t grayScale(const uchar4* const rbg_img, unsigned char *const grsc_img)
{
   
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    //cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    
    //cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    
    //cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    //cudaFree(dev_c);
    //cudaFree(dev_a);
    //cudaFree(dev_b);
    
    return cudaStatus;
}

//cpu implemenetation
//equation chosen for RGB to greyscale is Out = .299f * Red + .587f * Green + .114f * Blue. AKA luma transform
//or 			img1 				img2 				rowCount		columnCount
void RGB2GSCALE_CPU(const uchar4* const rbg_img, unsigned char *const grsc_img, const size_t rowCount, const size_t colCount) {

	size_t row, col;
	
	//brute force for loop iterating over all pixels in image and performing 		calculation
	for (row = 0; row<rowCount; ++row) {
		for (col = 0; col < colCount; ++col) {
			//displacement is total columns plus current column times current row
			int displacement = colCount + col * row;
			//obtain current working pixel
			const uchar4 colo = rbg_img[displacement]; //color can be represented using unsigned 4
			//apply formula for the greyscale rank
			const float chanSum = (.299f * colo.x) + (.587f * colo.y) + (.114f * colo.z); //these float values vary depending on what formula you use, we selected the most common
																			  //update greyscale image with the channel sum value
			grsc_img[displacement] = chanSum;


		}
	}


}

